/**
*Using a similar concept as question 2, you will now crack passwords using CUDA. As a kernel function 
*cannot use the crypt library, you will be given an encryption function instead which will 
*generate a password for you.  Your program will take in an encrypted password and decrypt it 
*using many threads on the GPU. CUDA allows multidimensional thread configurations so your kernel function 
*(which runs on the GPU) will need to be modified according to how you call your function. 
*
*Creatd by Nirmal Abeykoon Mudiynaslegae - 1811342
*6CS005 - High Performance Computing. 

***********************************************************************************************
					How to Run
*Compile as Normal nvcc - nvcc Password_cracking_cuda2.0.cu


*To run(need for command line arguments [blockx,blocky,threadx,thready] in this oder)
*this works with any number of threads upto 67600
*max blockx and blocky values are 26
*max Threadx and Thready values are 10
*there are some pre encrypted passwords in main that user can uncommnet to test the program

*eg:- ./a.out 26 26 10 10

***********************************************************************************************


*/
// includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//Crypt a password
__device__ 
char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

// cheack if two strings are matching
__device__ 
int isMatching(char* charOne, char* charTwo, int length) {
	int result = 1;
	for (int i = 0; i < length; i++) {
		if (charOne[i] != charTwo[i]) {
			result = 0;
			break;
		}
	}
	return result;
}

//GPU kernal
__global__ 
void crack(char * cryptPassword , char * GpuPPass){

	//close thread if password is already found
	if(*GpuPPass!= NULL){
		return;
	}
	
	int combos = 26*26*100;
	long threadCount = gridDim.x*gridDim.y*blockDim.x*blockDim.y;
	int remainder = combos % threadCount;
	int start;
	int end;
	
	int newCombos = combos-remainder;
	long workperthread = newCombos/threadCount;
	
	//create unique 2D thread ID
	int blockId = blockIdx.x+blockIdx.y*gridDim.x;
	int threadId = blockId*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x)+threadIdx.x;
	
	//create unique start and end point for each thread
	if (threadId == 0){
		start = 0;
		
	}else{
		start = threadId*workperthread;
		
	}
	if (threadId == (threadCount-1)){
		end = combos;
		//printf("ThreadID = %d Start=%d End=%d\n",threadId,start,end);
		//printf("%d\n",end);
	}else{
		end = (threadId*workperthread)+workperthread;
	}
	
	//printf("ThreadID = %d Start=%d End=%d\n",threadId,start,end);
	for(int i=start; i<end; i++){
		char  plain[5];
		char * TempcryptPassword;
		
		//close thread if password is already found
		if(*GpuPPass!= NULL){
			return;
		}
		
		//create unique password matching to uniqe thread number and workload
		int letterAndNumber = 26*100;
		int firstIndex = i/letterAndNumber;
		char firstChar = (char)( firstIndex + 'a');
		
		int secondAlphabetIndex = (i / 100) % 26;
		char secondChar = (char) (secondAlphabetIndex + 'a');
		int intNumbers= i % 100;
		char numberOne = intNumbers/10+'0';
		char numberTwo = intNumbers%10+'0';
		
		//printf("%c\n",(intNumbers%10+'0'));

		
		plain[0] = firstChar;
		plain[1] = secondChar;
		plain[2] = numberOne ;
		plain[3] = numberTwo;
			
		//temp crypt password
		//printf("Cheacked PassWord--->%s\n", plain);
		TempcryptPassword = CudaCrypt(plain);
		
		//if password matches save the results
		if ( isMatching(cryptPassword, TempcryptPassword, 11) > 0 )
		{
			//printf("GPU found the password PassWord--->%s\n", plain);
			for(int i=0; i < 4; i++){
				GpuPPass[i]=plain[i];	
			}
			
			//close thread if password is already found
			if(*GpuPPass!= NULL){
				return;
			}
		}
		
	}

}

//main
int main(int argc, char ** argv){

	char* CpuPPass;

	/*Some passwords for testing*/
	//const char cryptPassword [] = "ccbddb2244"; //aa00
	//const char cryptPassword [] = "cxbdwy2745"; //zz99
	const char cryptPassword [] = "plosmo2723"; //np97
	//const char cryptPassword [] = "hdgwqs7380"; //ft54
	
	if (argc < 5) {
		printf("Usage: %s need 4 args\n",argv[0]);
		exit(-1);
	}
	
	
	//thread and block cound from user
	int blockx = atoll(argv[1]);
	int blocky = atoll(argv[2]);
	int threadx = atoll(argv[3]);
	int thready = atoll(argv[4]);
		
	if ((blockx > 26) || (blocky > 26) ){
		printf("Usage: Blockx and Blocky values Must be less than 26\n");
		exit(-1);
	}
	
	if ((threadx > 10) || (thready > 10) ){
		printf("Usage: Threadx and Thready values Must be less than 10\n");
		exit(-1);
	}

	
	char * gpuCryptPassword;
	hipMalloc((void**)&gpuCryptPassword,sizeof(cryptPassword));
	hipMemcpy(gpuCryptPassword,cryptPassword,sizeof(cryptPassword),hipMemcpyHostToDevice);
	
	char *GpuPPass;
	hipMalloc((void**)&GpuPPass, sizeof(char)*5);

	crack<<< dim3(blockx,blocky,1), dim3(threadx,thready,1) >>>(gpuCryptPassword,GpuPPass);
	hipDeviceSynchronize();
	
	// Copy password from device to host
	CpuPPass = (char*)malloc( sizeof(char) * 5 );
	hipMemcpy(CpuPPass, GpuPPass, sizeof(char)*5, hipMemcpyDeviceToHost);
	
	
	//print if password was found
	if (CpuPPass != NULL && CpuPPass[0] != 0) {
		printf("Encrypted PassWord--->%s\n", cryptPassword);
		printf("Decrypted PassWord--->%s\n", CpuPPass);
	} else {
		printf("Unable to find the password.\n");
	}
	
	
	//free memory
	hipFree(gpuCryptPassword);
	hipFree(GpuPPass);
	free(CpuPPass);
	return 0;
}
